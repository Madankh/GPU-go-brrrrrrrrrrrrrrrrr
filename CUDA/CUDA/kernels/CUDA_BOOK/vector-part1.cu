#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
#define N 10000000
#define BLOCK_SIZE 250

void init_vector(float *vec, int n){
    for(int i=0; i<n; i++){
        vec[i] = (float)rand()/RAND_MAX;
    }
}

__global__ void vecAddcuda(float *d_A, float *d_B, float *d_C, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
        d_C[i] = d_A[i] + d_B[i];
    }
};

void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    for(int i=0;  i<n; i++){
        C_h[i] = A_h[i] + B_h[i];
    }
}

int main(){
    float *h_A, *h_B, *h_C, *h_gpu_result;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);
    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_gpu_result = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_A, N);
    init_vector(h_B, N);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size , hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size , hipMemcpyHostToDevice);

    // Define grid and block diminestion
    int num_blocks = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;

    // warm-up runs
    printf("Performing warm-up runs...\n");
    for(int i=0; i<3; i++){
        vecAdd(h_A, h_B, h_C, N);
    }

    printf("Performing CUDA warm-up runs...\n");
    for(int i=0; i<3; i++){
        vecAddcuda<<<num_blocks , BLOCK_SIZE>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
    }
    hipMemcpy(h_gpu_result, d_C, size, hipMemcpyDeviceToHost);

    // vecAdd(A, B, C, N);
    bool correct = true;
    for(int i=0; i<N; i++){
        if (fabs(h_C[i] - h_gpu_result[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(h_gpu_result);
    return 0;


    return 0;
}