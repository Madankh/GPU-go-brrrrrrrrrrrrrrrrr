#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
#define N 100000000 // vector SIZE = 10 Millions
#define BLOCK_SIZE 256

void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    for(int i=0;  i<n; i++){
        C_h[i] = A_h[i] + B_h[i];
    }
}
__global__ void vecAddCu(float* A_h, float* B_h , float *C_h, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
       C_h[i] = A_h[i] + B_h[i];
    }
}
void init_vector(float* vec , int n){
    for(int i=0; i<n; i++){
        vec[i] = (float)rand() / RAND_MAX;
    }
}

int main(){
    float *h_A, *h_B, *h_C, *h_C_gpu;
    size_t size = N * sizeof(float);
    
    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_C_gpu = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_A, N);
    init_vector(h_B, N);

    float *d_A, *d_B, *d_C;
    // allocate device mem
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and  block diminestion
    int num_blocks = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;

    // warm-up runs
    printf("Performing warm-up runs...\n");
    for(int i=0; i<3; i++){
        vecAdd(h_A, h_B, h_C, N);
    }

    printf("Performing CUDA warm-up runs...\n");
    for(int i=0; i<3; i++){
        vecAddCu<<<num_blocks, BLOCK_SIZE>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
    }
    hipMemcpy(h_C_gpu, d_C, size, hipMemcpyDeviceToHost);

    // vecAdd(A, B, C, N);
    bool correct = true;
    for(int i=0; i<3; i++){
        if (fabs(h_C[i] - h_C_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}